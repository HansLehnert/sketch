#include "hip/hip_runtime.h"
/**
 * @brief Countmin-CU sketch
 *
 * CUDA implementation
 *
 * @file sketch.cpp
 * @author Hans Lehnert
 */

#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <chrono>
#include <limits>
#include <thread>
#include <future>
#include <unordered_map>

#include "fasta.hpp"
#include "MappedFile.hpp"


const unsigned int MAX_LENGTH = 28;

const unsigned int N_HASH = 4;
const unsigned int HASH_BITS = 14;

// Seeds
__constant__ uint16_t d_seeds[N_HASH * MAX_LENGTH * 2];


struct SketchSettings {
    int min_length;
    int max_length;
    int n_length;

    std::vector<int> threshold;

    float growth;
};


struct Sketch {
    int32_t count[N_HASH][1 << HASH_BITS];
};


/**
 * @brief
 * Compute H3 hash
 *
 * Compute the H3 hash on a set of keys using constant memory seeds. Keys are
 * shifted by the offset, to start the hash.
 */
template <int n_hash>
__global__ void hashH3(
        int n,
        int bits,
        uint64_t* keys,
        uint16_t* src,
        uint16_t* dst,
        int offset) {
    unsigned int start_index = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int stride = blockDim.x * gridDim.x;

    for (unsigned int i = start_index; i < n; i += stride) {
        for (int j = 0; j < n_hash; j++)
            dst[i * n_hash + j] = src[i * n_hash + j];

        unsigned long key = keys[i] >> offset;
        for (int j = 0; j < bits; j++) {
            if (key & 1) {
                for (int k = 0; k < n_hash; k++) {
                    dst[i * n_hash + k] ^=
                        d_seeds[(j + offset) * n_hash + k];
                }
            }
            key >>= 1;
        }
    }
}


void sketchWorker(
        const SketchSettings& settings,
        int start,
        int stride,
        const uint16_t* d_hashes,
        const std::vector<unsigned long>& test_data,
        const std::vector<unsigned long>& control_data,
        const std::vector<unsigned char>& test_lengths,
        const std::vector<unsigned char>& control_lengths,
        std::vector<std::unordered_map<uint64_t, int>>* heavy_hitters_vec) {

    int test_data_size = test_data.size();
    uint16_t* h_hashes = new uint16_t[N_HASH * test_data_size];

    for (int n = start; n < settings.n_length; n += stride) {
        uint64_t mask = ~(~0UL << ((settings.min_length + n) * 2));

        Sketch sketch = {0};
        int length = settings.min_length + n;

        std::unordered_map<uint64_t, int> heavy_hitters;

        // Copy hashes from device
        hipMemcpy(
            h_hashes,
            &d_hashes[N_HASH * test_data_size * n],
            N_HASH * test_data_size * sizeof(uint16_t),
            hipMemcpyDeviceToHost
        );

        // Hash values
        for (int i = 0; i < test_data_size; i++) {
            if (test_lengths[i] < length)
                continue;

            int min_hits = std::numeric_limits<int>::max();
            uint16_t* hashes = &h_hashes[i * N_HASH];

            for (int j = 0; j < N_HASH; j++) {
                if (sketch.count[j][hashes[j]] < min_hits) {
                    min_hits = sketch.count[j][hashes[j]];
                }
            }

            for (int j = 0; j < N_HASH; j++) {
                if (sketch.count[j][hashes[j]] == min_hits) {
                    sketch.count[j][hashes[j]]++;
                }
            }

            min_hits++;

            if (min_hits >= settings.threshold[n]) {
                uint64_t sequence = test_data[i] & mask;
                heavy_hitters[sequence] = min_hits;
            }
        }

        for (auto& i : heavy_hitters) {
            i.second /= settings.growth;
        }

        // Control step
        for (int i = 0; i < control_data.size(); i++) {
            if (control_lengths[i] < length)
                continue;

            std::unordered_map<uint64_t, int>::iterator counter;
            counter = heavy_hitters.find(control_data[i] & mask);
            if (counter != heavy_hitters.end()) {
                counter->second--;
            }
        }

        // Select only the heavy-hitters not in the control set
        auto i = heavy_hitters.begin();
        while (i != heavy_hitters.end()) {
            if (i->second <= 0) {
                i = heavy_hitters.erase(i);
            }
            else {
                i++;
            }
        }

        (*heavy_hitters_vec)[n] = heavy_hitters;
    }

    delete[] h_hashes;
}


int main(int argc, char* argv[]) {
    if (argc < 5) {
        std::cerr
            << "Usage:" << std::endl
            << '\t' << argv[0]
            << " test_set control_set min_length max_length threshold_1 ..."
            << std::endl;
        return 1;
    }

    // Configure sketch settings
    SketchSettings settings;
    settings.min_length = atoi(argv[3]);
    settings.max_length = atoi(argv[4]);
    settings.n_length = settings.max_length - settings.min_length + 1;
    settings.growth = 2.0;

    if (argc - 5 < settings.n_length) {
        std::cerr
            << "Missing threshold values. Got "
            << argc - 5
            << ", expected "
            << settings.n_length
            << std::endl;
        return 1;
    }

    for (int i = 5; i < argc; i++) {
        settings.threshold.push_back(atoi(argv[i]));
    }

    // Generate seeds
    uint16_t* h_seeds = new uint16_t[N_HASH * MAX_LENGTH * 2];
    for (int i = 0; i < N_HASH * MAX_LENGTH * 2; i++)
        h_seeds[i] = rand() & ~(~0UL << HASH_BITS);
    hipMemcpyToSymbol(HIP_SYMBOL(d_seeds), h_seeds, sizeof(d_seeds));
    hipDeviceSynchronize();
    delete[] h_seeds;

    // Load memory mapped files
    MappedFile test_file = MappedFile::load(argv[1]);
    MappedFile control_file = MappedFile::load(argv[2]);

    // Heavy-hitters containers
    std::vector<std::unordered_map<uint64_t, int>> heavy_hitters;
    heavy_hitters.resize(settings.n_length);

    // Start time measurement
    auto start_time = std::chrono::steady_clock::now();

    // Parse data set and transfer to device
    std::vector<unsigned char> test_lengths;
    std::vector<unsigned long> test_data = parseFasta(
        test_file.data(),
        test_file.size(),
        settings.min_length,
        ~(~0UL << (settings.max_length * 2)),
        &test_lengths);

    unsigned long n_data_test = test_data.size();
    unsigned long* d_data_test;

    auto preprocessing_time = std::chrono::steady_clock::now();

    // Copy sequences to device
    hipMalloc(&d_data_test, n_data_test * sizeof(unsigned long));
    hipMemcpyAsync(
        d_data_test,
        test_data.data(),
        n_data_test * sizeof(unsigned long),
        hipMemcpyHostToDevice);

    // Allocate memory for hashes and sketches
    uint16_t* d_hashes;
    size_t hash_data_size =
        settings.n_length * n_data_test * N_HASH * sizeof(uint16_t);

    hipMalloc(&d_hashes, hash_data_size);

    // Calculate hashes for the first length.
    // The first is a special case since it needs to hash over MIN_LENGTH
    // symbols instead of only one
    int block_size = 256;
    int num_blocks = 16;

    hashH3<N_HASH><<<block_size, num_blocks, 0>>>(
        n_data_test,
        settings.min_length * 2,
        d_data_test,
        &d_hashes[0],
        &d_hashes[0],
        0);

    // Compute for the rest of the k-mers lengths
    for (int i = 1; i < settings.n_length; i++) {
        hashH3<N_HASH><<<num_blocks, block_size, 0>>>(
            n_data_test,
            2,
            d_data_test,
            &d_hashes[n_data_test * N_HASH * (i - 1)],
            &d_hashes[n_data_test * N_HASH * i],
            (settings.min_length + i - 1) * 2);
    }

    // Sync device in separate thread to measure total hashing time
    std::chrono::time_point<std::chrono::steady_clock> hash_time;
    auto cuda_sync = std::async(
        [&] {
            hipDeviceSynchronize();
            hash_time = std::chrono::steady_clock::now();
        }
    );

    // Parse control file during hash calculation
    std::vector<unsigned char> control_lengths;
    std::vector<unsigned long> control_data = parseFasta(
        control_file.data(),
        control_file.size(),
        settings.min_length * 2,
        ~(~0UL << (settings.max_length * 2)),
        &control_lengths);

    cuda_sync.wait();

    // Create threads
    int n_threads = std::thread::hardware_concurrency();
    std::vector<std::thread> threads;
    threads.reserve(n_threads);

    for (int i = 0; i < n_threads; i++) {
        threads.emplace_back(
            sketchWorker,
            settings,
            i,
            n_threads,
            d_hashes,
            test_data,
            control_data,
            test_lengths,
            control_lengths,
            &heavy_hitters
        );
    }

    for (int i = 0; i < threads.size(); i++) {
        threads[i].join();
    }

    // End time measurement
    auto end_time = std::chrono::steady_clock::now();
    std::chrono::duration<double> preprocessing_diff = preprocessing_time - start_time;
    std::chrono::duration<double> hash_diff = hash_time - preprocessing_time;
    std::chrono::duration<double> total_diff = end_time - start_time;

    std::clog << "Preprocessing time: " << preprocessing_diff.count() << " s" << std::endl;
    std::clog << "Hashing time: " << hash_diff.count() << " s" << std::endl;
    std::clog << "Execution time: " << total_diff.count() << " s" << std::endl;
    std::clog << "Data vectors: " << n_data_test << std::endl;

    // Print heavy-hitters
    int heavy_hitters_count = 0;

    for (int n = 0; n < settings.n_length; n++) {
        heavy_hitters_count += heavy_hitters[n].size();
        std::clog
            << "Heavy-hitters (length " << settings.min_length + n << "): "
            << heavy_hitters[n].size() << std::endl;

        for (auto x : heavy_hitters[n]) {
            std::cout
                << sequenceToString(x.first, settings.min_length + n)
                << std::endl;
        }
    }

    std::clog << "Heavy-hitters (total): " << heavy_hitters_count << std::endl;

    // Free shared memory
    // hipFree(d_data);
    // hipFree(d_sketch);
    // hipFree(d_hashes);
    // hipFree(d_heavy_hitters);
    // hipFree(heavy_hitters_count);
    // hipHostFree(h_data);

    return 0;
}
